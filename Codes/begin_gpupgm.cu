#include <hip/hip_runtime.h>

#include <vector>
#include <algorithm>
#include <random>
#include <iostream>

// Assuming ear_premble is defined elsewhere (e.g., in a header)
struct ear_premble {
    int start_node;
    int end_node;
    std::vector<int> nodes;
    // Add other fields as needed
};

// Kernel declarations
__global__ void bc_kernel_active(int* R, int* C, int* F, int g_n, int g_m, float* bc, int* sampled_nodes, int sample_size, ear_premble* ear_data);
__global__ void bc_kernel_free(int* R, int* C, int* F, int g_n, int g_m, float* bc, int* sampled_nodes, int sample_size, ear_premble* ear_data);

// First begin_gpu overload (called at line 165)
void begin_gpu(int*& R, int*& C, int*& F, int g_n, int g_m, ear_premble**& ear_active, ear_premble**& ear_free,
               std::vector<int>& active_nodes, std::vector<int>& free_nodes, float*& h_bc, double& gpu_time) {
    // Start timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // Determine sample size
    int sample_size = static_cast<int>(std::sqrt(static_cast<float>(g_n)));
    if (sample_size < 1) sample_size = 1;

    // Sample nodes (prefer active nodes, fallback to all nodes)
    std::vector<int> nodes = active_nodes.empty() ? std::vector<int>(g_n) : active_nodes;
    if (nodes.size() == g_n) {
        for (int i = 0; i < g_n; ++i) nodes[i] = i;
    }
    std::random_device rd;
    std::mt19937 g(rd());
    std::shuffle(nodes.begin(), nodes.end(), g);
    if (sample_size > nodes.size()) sample_size = nodes.size();
    std::vector<int> sampled_nodes(nodes.begin(), nodes.begin() + sample_size);

    // Device memory
    int *d_R, *d_C, *d_F, *d_sampled_nodes;
    float* d_bc;
    ear_premble *d_ear_active, *d_ear_free;

    // Allocate device memory
    hipMalloc(&d_R, g_m * sizeof(int));
    hipMalloc(&d_C, g_m * sizeof(int));
    hipMalloc(&d_F, (g_n + 1) * sizeof(int));
    hipMalloc(&d_bc, g_n * sizeof(float));
    hipMalloc(&d_sampled_nodes, sample_size * sizeof(int));
    hipMalloc(&d_ear_active, sizeof(ear_premble) * active_nodes.size());
    hipMalloc(&d_ear_free, sizeof(ear_premble) * free_nodes.size());

    // Copy data to device
    hipMemcpy(d_R, R, g_m * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_C, C, g_m * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_F, F, (g_n + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemset(d_bc, 0, g_n * sizeof(float));
    hipMemcpy(d_sampled_nodes, sampled_nodes.data(), sample_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_ear_active, *ear_active, sizeof(ear_premble) * active_nodes.size(), hipMemcpyHostToDevice);
    hipMemcpy(d_ear_free, *ear_free, sizeof(ear_premble) * free_nodes.size(), hipMemcpyHostToDevice);

    // Kernel launch
    int threadsPerBlock = 256;
    int blocks = (sample_size + threadsPerBlock - 1) / threadsPerBlock;
    bc_kernel_active<<<blocks, threadsPerBlock>>>(d_R, d_C, d_F, g_n, g_m, d_bc, d_sampled_nodes, sample_size, d_ear_active);
    hipDeviceSynchronize();
    bc_kernel_free<<<blocks, threadsPerBlock>>>(d_R, d_C, d_F, g_n, g_m, d_bc, d_sampled_nodes, sample_size, d_ear_free);
    hipDeviceSynchronize();

    // Copy results back
    hipMemcpy(h_bc, d_bc, g_n * sizeof(float), hipMemcpyDeviceToHost);

    // Scale results
    float scaling_factor = static_cast<float>(g_n) / sample_size;
    for (int i = 0; i < g_n; ++i) {
        h_bc[i] *= scaling_factor;
    }

    // Record time
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    gpu_time = milliseconds / 1000.0;

    // Free memory
    hipFree(d_R);
    hipFree(d_C);
    hipFree(d_F);
    hipFree(d_bc);
    hipFree(d_sampled_nodes);
    hipFree(d_ear_active);
    hipFree(d_ear_free);
}

// Second begin_gpu overload (called at line 174)
void begin_gpu(int*& R, int*& C, int*& F, int g_n, int g_m, ear_premble**& ear_active, std::vector<int>& active_nodes,
               std::vector<int>& free_nodes, std::vector<int>& ear_active_nodes, std::vector<int>& ear_free_nodes,
               int ear_count, std::vector<int>& ear_nodes, float*& h_bc, double& gpu_time, int*& ear_start, int*& ear_end) {
    // Start timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // Determine sample size
    int sample_size = static_cast<int>(std::sqrt(static_cast<float>(g_n)));
    if (sample_size < 1) sample_size = 1;

    // Sample nodes
    std::vector<int> nodes = active_nodes.empty() ? std::vector<int>(g_n) : active_nodes;
    if (nodes.size() == g_n) {
        for (int i = 0; i < g_n; ++i) nodes[i] = i;
    }
    std::random_device rd;
    std::mt19937 g(rd());
    std::shuffle(nodes.begin(), nodes.end(), g);
    if (sample_size > nodes.size()) sample_size = nodes.size();
    std::vector<int> sampled_nodes(nodes.begin(), nodes.begin() + sample_size);

    // Device memory
    int *d_R, *d_C, *d_F, *d_sampled_nodes, *d_ear_start, *d_ear_end;
    float* d_bc;
    ear_premble* d_ear_active;

    // Allocate device memory
    hipMalloc(&d_R, g_m * sizeof(int));
    hipMalloc(&d_C, g_m * sizeof(int));
    hipMalloc(&d_F, (g_n + 1) * sizeof(int));
    hipMalloc(&d_bc, g_n * sizeof(float));
    hipMalloc(&d_sampled_nodes, sample_size * sizeof(int));
    hipMalloc(&d_ear_active, sizeof(ear_premble) * ear_count);
    hipMalloc(&d_ear_start, ear_count * sizeof(int));
    hipMalloc(&d_ear_end, ear_count * sizeof(int));

    // Copy data to device
    hipMemcpy(d_R, R, g_m * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_C, C, g_m * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_F, F, (g_n + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemset(d_bc, 0, g_n * sizeof(float));
    hipMemcpy(d_sampled_nodes, sampled_nodes.data(), sample_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_ear_active, *ear_active, sizeof(ear_premble) * ear_count, hipMemcpyHostToDevice);
    hipMemcpy(d_ear_start, ear_start, ear_count * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_ear_end, ear_end, ear_count * sizeof(int), hipMemcpyHostToDevice);

    // Kernel launch
    int threadsPerBlock = 256;
    int blocks = (sample_size + threadsPerBlock - 1) / threadsPerBlock;
    bc_kernel_active<<<blocks, threadsPerBlock>>>(d_R, d_C, d_F, g_n, g_m, d_bc, d_sampled_nodes, sample_size, d_ear_active);
    hipDeviceSynchronize();
    // Note: Free nodes may be handled differently based on ear_nodes
    bc_kernel_free<<<blocks, threadsPerBlock>>>(d_R, d_C, d_F, g_n, g_m, d_bc, d_sampled_nodes, sample_size, d_ear_active);
    hipDeviceSynchronize();

    // Copy results back
    hipMemcpy(h_bc, d_bc, g_n * sizeof(float), hipMemcpyDeviceToHost);

    // Scale results
    float scaling_factor = static_cast<float>(g_n) / sample_size;
    for (int i = 0; i < g_n; ++i) {
        h_bc[i] *= scaling_factor;
    }

    // Record time
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    gpu_time = milliseconds / 1000.0;

    // Free memory
    hipFree(d_R);
    hipFree(d_C);
    hipFree(d_F);
    hipFree(d_bc);
    hipFree(d_sampled_nodes);
    hipFree(d_ear_active);
    hipFree(d_ear_start);
    hipFree(d_ear_end);
}

// Error checking utility
void checkCudaError(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        std::cerr << msg << ": " << hipGetErrorString(err) << std::endl;
        exit(1);
    }
}